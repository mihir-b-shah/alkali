
#include <hip/hip_runtime.h>
// RUN: cgeist --target aarch64-unknown-linux-gnu %s -nocudalib -nocudainc %stdinclude -S -o - | FileCheck %s -check-prefix=MLIR
// RUN: cgeist --target aarch64-unknown-linux-gnu %s -nocudalib -nocudainc %stdinclude -emit-llvm -S -o - | FileCheck %s -check-prefix=LLVM

// MLIR:  llvm.target_triple = "aarch64-unknown-linux-gnu"
// LLVM:  target triple = "aarch64-unknown-linux-gnu"

int main() { return 0; }
